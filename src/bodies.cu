#include "hip/hip_runtime.h"
/***************************************************************************//**
 * \file bodies.cu
 * \author Anush Krishnan (anush@bu.edu)
 * \author Christopher Minar (minarc@oregonstate.edu)
 * \brief Implementation of the methods of the class \c bodies.
 */


#include "bodies.h"
#include <cusp/blas/blas.h>
#include <iomanip>
#include <fstream>

/**
 * \brief Sets initial position and velocity of each body.
 *
 * \param db database that contains all the simulation parameters
 * \param D information about the computational grid
 */
void bodies::initialise(parameterDB &db, domain &D)
{
	std::cout << "Initialising bodies... ";
	std::vector<body> *B = db["flow"]["bodies"].get<std::vector<body> *>();

	// number of bodies in the flow
	numBodies = B->size();

	// set the sizes of all the arrays
	numPoints.resize(numBodies);
	offsets.resize(numBodies);

	startI.resize(numBodies);
	startJ.resize(numBodies);
	numCellsX.resize(numBodies);
	numCellsY.resize(numBodies);
	startI0.resize(numBodies);
	startJ0.resize(numBodies);
	numCellsX0.resize(numBodies);
	numCellsY0.resize(numBodies);

	xmin.resize(numBodies);
	xmax.resize(numBodies);
	ymin.resize(numBodies);
	ymax.resize(numBodies);
	xmin0.resize(numBodies);
	xmax0.resize(numBodies);
	ymin0.resize(numBodies);
	ymax0.resize(numBodies);
	

	// calculate offsets, number of points in each body and the total number of points
	totalPoints = 0;
	for(int k=0; k<numBodies; k++)
	{
		offsets[k] = totalPoints;
		numPoints[k] = (*B)[k].numPoints;
		totalPoints += numPoints[k];
	}

	// fill up coordinates of body points
	X.resize(totalPoints);
	Y.resize(totalPoints);
	ds.resize(totalPoints);
	ones.resize(totalPoints);
	cusp::blas::fill(ones, 1.0);
	for(int k=0; k<numBodies; k++)
	{
		for(int i=0; i<numPoints[k]; i++)
		{
			X[i+offsets[k]] = (*B)[k].X[i];
			Y[i+offsets[k]] = (*B)[k].Y[i];
		}
	}
	x.resize(totalPoints);
	y.resize(totalPoints);
	uB.resize(totalPoints);
	vB.resize(totalPoints);
	uBk.resize(totalPoints);
	vBk.resize(totalPoints);
	I.resize(totalPoints);
	J.resize(totalPoints);

	force_pressure.resize(totalPoints);
	x1.resize(totalPoints);
	x2.resize(totalPoints);
	x3.resize(totalPoints);
	x4.resize(totalPoints);
	y1.resize(totalPoints);
	y2.resize(totalPoints);
	y3.resize(totalPoints);
	y4.resize(totalPoints);
	q1.resize(totalPoints);
	q2.resize(totalPoints);
	q3.resize(totalPoints);
	q4.resize(totalPoints);
	point_y.resize(totalPoints);
	point_x.resize(totalPoints);
	point2_y.resize(totalPoints);
	point2_x.resize(totalPoints);
	point3_y.resize(totalPoints);
	point3_x.resize(totalPoints);
	centerVelocityU = 0;
	centerVelocityV = 0;

	cusp::blas::fill(vB, 0);
	cusp::blas::fill(uB, 0);
	cusp::blas::fill(vBk, 0);
	cusp::blas::fill(uBk, 0);

	bodiesMove = false;
	for(int k=0; k<numBodies; k++)
	{
		// assume a closed body (closed loop)
		for(int i=offsets[k], j = offsets[k]+numPoints[k]-1; i<offsets[k]+numPoints[k];)
		{
			// calculate the lengths of the boundary segments
			ds[i] = sqrt( (X[i]-X[j])*(X[i]-X[j]) + (Y[i]-Y[j])*(Y[i]-Y[j]) );

			// j takes the value of i, then i is incremented
			j = i++;
		}
		// if the body is moving, set bodiesMove to true
		bodiesMove = bodiesMove || (*B)[k].moving[0] || (*B)[k].moving[1];
	}
	// set initial position of the body
	update(db, D, 0.0);

	if(numBodies)
	{
		calculateCellIndices(D);
		calculateTightBoundingBoxes(db, D);
		calculateBoundingBoxes(db, D);
	}

	midX=0;
	midY=0;
	midX0=0;
	midY0=0;
	for (int i=0;i<totalPoints;i++)
	{
		midX += x[i];
		midY += y[i];
	}
	midX /= totalPoints;
	midY /= totalPoints;
	midX=midX0;
	midY=midY0;
	centerVelocityV = 0;
	centerVelocityU = 0;
	centerVelocityU0= 0;
	centerVelocityV0= 0;
}

/**
 * \brief Stores index of each cell that contains a boundary point.
 *
 * It calculates the index of the x-coordinate and the index of the y-coordinate
 * of the bottom-left node of each cell that contains a boundary point.
 * This information is useful when transferring data between the boundary points
 * and the computational grid.
 *
 * \param D information about the computational grid
 */

void bodies::calculateCellIndices(domain &D)
{
	int	i=0, j=0;

	// find the cell for the zeroth point
	while(D.x[i+1] < x[0])
		i++;
	while(D.y[j+1] < y[0])
		j++;
	I[0] = i;
	J[0] = j;

	for(int k=1; k<totalPoints; k++)
	{
		// if the next boundary point is to the left of the current boundary point
		if(x[k] < x[k-1])
		{
			while(D.x[i] > x[k])
				i--;
		}
		// if the next boundary point is to the right of the current boundary point
		else
		{
			while(D.x[i+1] < x[k])
				i++;
		}
		// if the next boundary point is below the current boundary point
		if(y[k] < y[k-1])
		{
			while(D.y[j] > y[k])
				j--;
		}
		// if the next boundary point is above the current boundary point
		else
		{
			while(D.y[j+1] < y[k])
				j++;
		}
		I[k] = i;
		J[k] = j;
	}
}

/**
 * \brief Calculates indices of the bounding box of each body in the flow.
 *
 * First the bounding box is scaled by a coefficient stored in the database.
 * Then, indices of the x-coordinate and y-coordinate of the bottom left cell
 * of the bounding box are stored. Finally, the number of cells in the x- and y-
 * directions are calculated.
 *
 * \param db database that contains all the simulation parameters
 * \param D information about the computational grid
 */
void bodies::calculateBoundingBoxes(parameterDB &db, domain &D)
{
	double scale = db["simulation"]["scaleCV"].get<double>(),
	     dx, dy;
	int  i, j;
	for(int k=0; k<numBodies; k++)
	{
		xmin[k] = x[offsets[k]];
		xmax[k] = xmin[k];
		ymin[k] = y[offsets[k]];
		ymax[k] = ymin[k];
		for(int l=offsets[k]+1; l<offsets[k]+numPoints[k]; l++)
		{
			if(x[l] < xmin[k]) xmin[k] = x[l];
			if(x[l] > xmax[k]) xmax[k] = x[l];
			if(y[l] < ymin[k]) ymin[k] = y[l];
			if(y[l] > ymax[k]) ymax[k] = y[l];
		}
		dx = xmax[k]-xmin[k];
		dy = ymax[k]-ymin[k];
		xmax[k] += 0.5*dx*(scale-1.0);
		xmin[k] -= 0.5*dx*(scale-1.0);
		ymax[k] += 0.5*dy*(scale-1.0);
		ymin[k] -= 0.5*dy*(scale-1.0);
		
		i=0; j=0;
		while(D.x[i+1] < xmin[k])
			i++;
		while(D.y[j+1] < ymin[k])
			j++;
		startI[k] = i;
		startJ[k] = j;
		
		while(D.x[i] < xmax[k])
			i++;
		while(D.y[j] < ymax[k])
			j++;
		numCellsX[k] = i - startI[k];
		numCellsY[k] = j - startJ[k];
	}
}

void bodies::calculateTightBoundingBoxes(parameterDB &db, domain &D)
{
	double scale = db["simulation"]["scaleCV"].get<double>();
	int  i, j;
	for(int k=0; k<numBodies; k++)
	{
		xmin0[k] = x[offsets[k]];
		xmax0[k] = xmin[k];
		ymin0[k] = y[offsets[k]];
		ymax0[k] = ymin[k];
		for(int l=offsets[k]+1; l<offsets[k]+numPoints[k]; l++)
		{
			if(x[l] < xmin0[k]) xmin0[k] = x[l];
			if(x[l] > xmax0[k]) xmax0[k] = x[l];
			if(y[l] < ymin0[k]) ymin0[k] = y[l];
			if(y[l] > ymax0[k]) ymax0[k] = y[l];
		}

		i=0; j=0;
		while(D.x[i+1] < xmin0[k])
			i++;
		while(D.y[j+1] < ymin0[k])
			j++;
		startI0[k] = i;
		startJ0[k] = j;

		while(D.x[i] < xmax[k])
			i++;
		while(D.y[j] < ymax[k])
			j++;
		numCellsX0[k] = i - startI0[k];
		numCellsY0[k] = j - startJ0[k];
	}
}

/**
 * \brief Updates position, velocity and neighbors of each body.
 *
 * This is done using the formulae:
 *
 * \f$ x_{i,m} = X^c_m + (X_{i,m} - X^0_m) \cos\theta - (Y_{i,m} - Y^0_m) \sin\theta \f$
 *
 * and
 *
 * \f$ y_{i,m} = Y^c_m + (X_{i,m} - X^0_m) \sin\theta + (Y_{i,m} - Y^0_m) \cos\theta \f$
 *
 * \param db database that contains all the simulation parameters
 * \param D information about the computational grid
 * \param Time the time
 */
void bodies::update(parameterDB &db, domain &D, double Time)
{
	typedef typename cusp::array1d<double, cusp::device_memory> Array;
	typedef typename Array::iterator                 Iterator;
	typedef cusp::array1d_view<Iterator>             View;

	// views of the vectors that store the coordinates and velocities of all the body points
	View    XView, YView, xView, yView, onesView, uBView, vBView;

	// body data
	std::vector<body> *B = db["flow"]["bodies"].get<std::vector<body> *>();

	for(int l=0; l<numBodies; l++)
	{
		// update the location and velocity of the body
		(*B)[l].update(Time);

		// create the views for the current body
		if(l < numBodies-1)
		{
			XView    = View(X.begin()+offsets[l], X.begin()+offsets[l+1]);
			YView    = View(Y.begin()+offsets[l], Y.begin()+offsets[l+1]);
			onesView = View(ones.begin()+offsets[l], ones.begin()+offsets[l+1]);
			uBView   = View(uB.begin()+offsets[l], uB.begin()+offsets[l+1]);
			vBView   = View(vB.begin()+offsets[l], vB.begin()+offsets[l+1]);
			xView    = View(x.begin()+offsets[l], x.begin()+offsets[l+1]);
			yView    = View(y.begin()+offsets[l], y.begin()+offsets[l+1]);
		}
		else
		{
			XView    = View(X.begin()+offsets[l], X.end());
			YView    = View(Y.begin()+offsets[l], Y.end());
			onesView = View(ones.begin()+offsets[l], ones.end());
			xView    = View(x.begin()+offsets[l], x.end());
			yView    = View(y.begin()+offsets[l], y.end());
			uBView   = View(uB.begin()+offsets[l], uB.end());
			vBView   = View(vB.begin()+offsets[l], vB.end());

		}

		// update postitions
		// x-coordinates
		cusp::blas::axpbypcz( onesView, XView, onesView, xView, (*B)[l].Xc[0],  cos((*B)[l].Theta), -(*B)[l].X0[0]*cos((*B)[l].Theta) );
		cusp::blas::axpbypcz( xView,    YView, onesView, xView,           1.0, -sin((*B)[l].Theta),  (*B)[l].X0[1]*sin((*B)[l].Theta) );
		// y-coordinates
		cusp::blas::axpbypcz( onesView, XView, onesView, yView, (*B)[l].Xc[1],  sin((*B)[l].Theta), -(*B)[l].X0[0]*sin((*B)[l].Theta) );
		cusp::blas::axpbypcz( yView,    YView, onesView, yView,           1.0,  cos((*B)[l].Theta), -(*B)[l].X0[1]*cos((*B)[l].Theta) );

		// update velocities
		// x-velocities
		cusp::blas::axpbypcz(onesView, yView, onesView, uBView, (*B)[l].vel[0], -(*B)[l].angVel,  (*B)[l].angVel*(*B)[l].Xc[1]);
		// y-velocities
		cusp::blas::axpbypcz(onesView, xView, onesView, vBView, (*B)[l].vel[1],  (*B)[l].angVel, -(*B)[l].angVel*(*B)[l].Xc[0]);
	}

	if(numBodies)
		calculateCellIndices(D);
}


/**
 * \brief Writes body coordinates into a file (using data from the device).
 *
 * \param caseFolder directory of the simulation
 * \param timeStep time-step of the simulation
 */
void bodies::writeToFile(std::string &caseFolder, int timeStep)
{
	cusp::array1d<double, cusp::host_memory>
		xHost = x,
		yHost = y;
	double *bx = thrust::raw_pointer_cast(&(xHost[0])),
	     *by = thrust::raw_pointer_cast(&(yHost[0]));
	writeToFile(bx, by, caseFolder, timeStep);
}

/**
 * \brief Writes body coordinates into a file called \a bodies.
 *
 * \param bx x-coordinate of all points of all bodies
 * \param by y-coordinate of all points of all bodies
 * \param caseFolder directory of the simulation
 * \param timeStep time-step of the simulation
 */
void bodies::writeToFile(double *bx, double *by, std::string &caseFolder, int timeStep)
{
	std::string       path;
	std::stringstream out;
	out << caseFolder << '/' << std::setfill('0') << std::setw(7) << timeStep << "/bodies";
	std::ofstream file(out.str().c_str());;
	file << '#' << std::setw(19) << "x-coordinate" << std::setw(20) << "y-coordinate" << std::endl;
	for (int l=0; l < totalPoints; l++)
	{
		file << bx[l] << '\t' << by[l] << '\n';
	}
	file.close();
}
