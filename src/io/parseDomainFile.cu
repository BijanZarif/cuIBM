#include "hip/hip_runtime.h"
/***************************************************************************//**
 * \file parseDomainFile.cu
 * \brief Parse the input file domain.yaml to obtain information about the
 *        computational grid.
 */


#include <fstream>
#include <yaml-cpp/yaml.h>
#include "io.h"

/**
 * \namespace io
 * \brief Contains functions related to I/O tasks.
 */
namespace io
{

using std::string;

/**
 * \brief Overloads the operator >>. Gets information from the parsed domain file.
 *
 * \param node the parsed file
 * \param D instance of the class \c domain to be filled
 */
void operator >> (const YAML::Node &node, domain &D)
{
	string dir;
	double start;
	int  numCells;
	
	node["direction"] >> dir;
	node["start"] >> start;

	if (dir=="x")
		D.nx = 0;
	else if(dir=="y")
		D.ny = 0;

	const YAML::Node &subDomains = node["subDomains"];
	for (unsigned int i=0; i<subDomains.size(); i++) //first pass, gets nx and ny and resizes the arrays appropriately
	{
		subDomains[i]["cells"] >> numCells;
		if (dir=="x")
			D.nx += numCells;
		else if(dir=="y")
			D.ny += numCells;
	}

	// allocate memory
	int  beg = 0;
	if(dir=="x")//x
	{
		D.x.resize(D.nx);	//x location of the pressure nodes (cell center)
		D.dx.resize(D.nx);	//x width of the pressure nodes
		D.xD.resize(D.nx);	//x location of the pressure nodes (cell center) on the device
		D.dxD.resize(D.nx);	//x width of the pressure nodes on the device
		D.xv.resize(D.nx);	//x location of where v is stored (same x as pressure node)
	}
	if(dir=="y")//y
	{
		D.y.resize(D.ny);	//y location of pressure nodes (cell center)
		D.dy.resize(D.ny);	//y height of pressure nodes
		D.yD.resize(D.ny);	//y location of pressure nodes (cell center) on the device
		D.dyD.resize(D.ny);	//y height of pressure nodes on the device
		D.yu.resize(D.ny);	//y location of where u is stored
	}

	// second pass, fills x,y,xv,yu,dx,dy
	double end, stretchRatio, h;
	for (unsigned int i=0; i<subDomains.size(); i++)
	{
		subDomains[i]["end"] >> end;
		subDomains[i]["cells"] >> numCells;
		subDomains[i]["stretchRatio"] >> stretchRatio;
		
		if(fabs(stretchRatio-1.0) < 1.0e-6)  //no cell stretching
		{
			h = (end - start)/numCells;
			for (int j=beg; j<beg+numCells; j++)
			{
				if (dir=="x")
				{
					if (j == beg)
					{
						D.x[j] = start + h/2; 	//start designates the boundary location so the location of the cell center is dx/2 away
						D.xv[j] = D.x[j];     	//xv is measured in the same x location as the cell center
						D.dx[j] = h;			//without stretching dx = h
					}
					else
					{
						D.dx[j] = h;			//without stretching dx = h
						D.x[j] = D.x[j-1] + 0.5*D.dx[j] + 0.5*D.dx[j-1];//dx is the width of the cell so to move over to the next cell center you need half of each cells dx
						D.xv[j] = D.x[j];		//x_xv = x_p
					}
				}
				else if (dir=="y")
				{
					if (j == beg)
					{
						D.y[j] = start + h/2;
						D.yu[j] = D.y[j];
						D.dy[j] = h;
					}
					else
					{
						D.dy[j] = h;
						D.y[j] = D.y[j-1] + 0.5*D.dy[j] + 0.5*D.dy[j-1];
						D.yu[j] = D.y[j];
					}
				}//end x/y elseif
			}//end for
		}//end no cell stretching
		else //cell stretching
		{
			h = (end - start)*(stretchRatio-1)/(pow(stretchRatio, numCells)-1); //the initial dx, will either be the largest or smallest value depending on if stretch is greater or less than 1
			for (int j=beg; j<beg+numCells; j++)
			{
				if (dir=="x")
				{
					if (j == beg)
					{
						D.x[j] = start + h*pow(stretchRatio, j-beg)/2;
						D.xv[j] = D.x[j];
						D.dx[j] = h*pow(stretchRatio, j-beg); //dx = biggest possible cell * stretch^(j-beg)
					}
					else
					{
						D.dx[j] = h*pow(stretchRatio, j-beg);
						D.x[j] = D.x[j-1] + 0.5*D.dx[j] + 0.5*D.dx[j-1];
						D.xv[j] = D.x[j];
					}
				}
				if (dir=="y")
				{
					if (j == beg)
					{
						D.y[j] = start + h*pow(stretchRatio, j-beg)/2;
						D.yu[j] = D.x[j];
						D.dy[j] = h*pow(stretchRatio, j-beg);
					}
					else
					{
						D.dy[j] = h*pow(stretchRatio, j-beg);
						D.y[j] = D.y[j-1] + 0.5*D.dy[j] + 0.5*D.dy[j-1];
						D.yu[j] = D.y[j];
					}
				}//end x/y elseif
			}//end for
		}//end cell stretching
		beg += numCells;
		start = end;
	}

	if(dir=="x")
	{
		D.xD  = D.x;
		D.dxD = D.dx;
	}
	else if(dir=="y")
	{
		D.yD  = D.y;
		D.dyD = D.dy;
	}
}

/**
 * \brief Parses the \a domain file and generates the computational grid.
 *
 * \param domFile the file that contains information about the computational grid
 * \param D instance of the class \c domain that will be filled with information about the computational grid
 */
void parseDomainFile(std::string &domFile, domain &D)
{
	std::ifstream fin(domFile.c_str());			//setup to go through casefolder/domain.yaml
	YAML::Parser  parser(fin);
	YAML::Node    doc;
	parser.GetNextDocument(doc);
	for (unsigned int i=0; i<doc.size(); i++)	//go through each node in domain.yaml
		doc[i] >> D;

	D.yv.resize(D.ny-1);	//resize variables, y location of where v is stored, offset dy/2 above cell center
	D.xu.resize(D.nx-1);	//x location of where u is stored, offset dx/2 right of cell center

	D.xuD.resize(D.nx-1);	//xu on device
	D.yuD.resize(D.ny);		//yu on device
	D.xvD.resize(D.nx);		//xv on device
	D.yvD.resize(D.ny-1);	//yv on device

	for(int i=0; i<D.nx-1; i++)	//set xu
	{
		D.xu[i] = D.x[i] + D.dx[i]/2;
	}
	
	for(int j=0; j<D.ny-1; j++)	//set yv
	{
		D.yv[j] = D.y[j] + D.dy[j]/2;
	}

	D.yD = D.y;		//set device variables to host variables
	D.xD = D.x;

	D.xuD = D.xu;
	D.yuD = D.yu;
	D.xvD = D.xv;
	D.yvD = D.yv;
}
} // end namespace io
