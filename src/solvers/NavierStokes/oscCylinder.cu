#include "hip/hip_runtime.h"
/***************************************************************************//**
 * \file  oscCylinder.cu
 * \author Christopher Minar (minarc@oregonstate.edu)
 * \based on code by Anush Krishnan (anush@bu.edu)
 * \brief Declaration of the class oscCylinder.
 */

#include <solvers/NavierStokes/oscCylinder/kernels/structure.h>
#include "oscCylinder.h"
#include <sys/stat.h>

/**
 * \brief Constructor. Copies the database and information about the computational grid.
 *
 * \param pDB database that contains all the simulation parameters
 * \param dInfo information related to the computational grid
 */
oscCylinder::oscCylinder(parameterDB *pDB, domain *dInfo)
{
	paramDB = pDB;
	domInfo = dInfo;
}

/**
 * \brief Writes data into files.
 */
void oscCylinder::writeData()
{
	parameterDB  &db = *paramDB;
	double dt  = db["simulation"]["dt"].get<double>();
	logger.startTimer("output");
	writeCommon();
	if (timeStep == 0)
		forceFile<<"timestep\tFx\tFxX\tFxY\tFxU\tFy\n";
	forceFile << timeStep*dt << '\t' << B.forceX << '\t'<<fxx<<"\t"<<fxy<<"\t"<<fxu<<"\t" << B.forceY << std::endl;//flag writing from b.forcex takes forever
	logger.stopTimer("output");
}

/**
 * \brief Writes numerical solution at current time-step,
 *        as well as the number of iterations performed in each solver,
 *        the force,
 *        and the middle position of the body (calculated as an average of all the nodes)
 */
void oscCylinder::writeCommon()
{
	luoIBM::writeCommon();
	midPositionFile << timeStep << '\t' << B.midX << '\t' << B.midY <<std::endl;
}

/*
 * Calculates new cell indices
 * Calculates new body bounding boxes
 * Tags Points
 * Remakes LHS matricies
 * updates Preconditioners
 */
void oscCylinder::updateSolver()
{
	B.calculateCellIndices(*domInfo);
	B.calculateBoundingBoxes(*paramDB, *domInfo);
	tagPoints();
	generateLHS1();//is this needed?
	generateLHS2();

	logger.startTimer("Preconditioner");
	if (iterationCount2 > 100)
	{
		PC.update(LHS1, LHS2);
	}
	logger.stopTimer("Preconditioner");
}

/*
 * Calculates Force
 * Moves body
 */
void oscCylinder::moveBody()
{
	parameterDB  &db = *paramDB;
	calculateForce();

	double *x_r	= thrust::raw_pointer_cast( &(B.x[0]) ),
		   *uB_r= thrust::raw_pointer_cast( &(B.uB[0]) );
	double	dt	= db["simulation"]["dt"].get<double>(),
			nu	= db["flow"]["nu"].get<double>(),
			t = dt*timeStep,
			f = 1,
			totalPoints=B.totalPoints,
			xold= B.midX,
			unew,
			xnew;

	xnew = -1/(2*M_PI)*sin(2*M_PI*f*t);
	unew = -f*cos(2*M_PI*f*t);

	B.centerVelocityU = unew;
	B.midX = xnew;

	const int blocksize = 256;
	dim3 grid( int( (totalPoints)/blocksize ) +1, 1);
	dim3 block(blocksize, 1);
	kernels::update_body_viv<<<grid,block>>>(x_r, uB_r, xnew-xold, unew, totalPoints);
}

/*
 * initialise the simulation
 */
void oscCylinder::initialise()
{
	luoIBM::initialise();

	//output
	parameterDB  &db = *paramDB;
	std::string folder = db["inputs"]["caseFolder"].get<std::string>();
	std::stringstream outPosition;
	outPosition << folder <<"/midPosition";
	midPositionFile.open(outPosition.str().c_str());

	double *x_r	= thrust::raw_pointer_cast( &(B.x[0]) ),
		   *uB_r= thrust::raw_pointer_cast( &(B.uB[0]) );
	double	dt	= db["simulation"]["dt"].get<double>(),
			nu	= db["flow"]["nu"].get<double>(),
			t = dt*timeStep,
			D = 0.2,
			uMax = 1,
			f = 1,
			KC = uMax/f/D,
			Re = uMax*D/nu,
			totalPoints=B.totalPoints,
			xold= B.midX,
			unew,
			xnew;

	std::cout<<"dt\tf\tuMax\tD\tnu\tRe\tKC\n";
	std::cout<<dt<<"\t"<<f<<"\t"<<uMax<<"\t"<<D<<"\t"<<nu<<"\t"<<Re<<"\t"<<KC<<"\n";
	xnew = -1/(2*M_PI)*sin(2*M_PI*f*t);
	unew = -f*cos(2*M_PI*f*t);
	B.centerVelocityU = unew;
	B.midX = xnew;

	const int blocksize = 256;
	dim3 grid( int( (totalPoints)/blocksize ) +1, 1);
	dim3 block(blocksize, 1);
	kernels::update_body_viv<<<grid,block>>>(x_r, uB_r, xnew-xold, unew, totalPoints);
}

/**
 * \brief Calculates the variables at the next time step.
 */
void oscCylinder::stepTime()
{
	generateRHS1();
	solveIntermediateVelocity();
	weightUhat();

	generateRHS2();
	solvePoisson();
	weightPressure();

	velocityProjection();
	//Release the body after a certain timestep
	if (timeStep >= (*paramDB)["simulation"]["startStep"].get<int>())
	{
		moveBody();
		updateSolver();
	}

	std::cout<<timeStep<<std::endl;
	timeStep++;

	if (timeStep%(*paramDB)["simulation"]["nsave"].get<int>() == 0)
	{
		//arrayprint(pressure,"p","p");
		//arrayprint(u,"u","x");
	}
}

/**
 * \brief Prints timing information and closes the different files.
 */
void oscCylinder::shutDown()
{
	luoIBM::shutDown();
	midPositionFile.close();
}

#include "oscCylinder/intermediateVelocity.inl"
