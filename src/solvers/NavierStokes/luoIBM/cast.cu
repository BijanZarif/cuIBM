#include "hip/hip_runtime.h"
#include <solvers/NavierStokes/luoIBM.h>

void luoIBM::cast()
{
	ghostTagsUV_r 						= thrust::raw_pointer_cast( &(ghostTagsUV[0]) );
	ghostTagsP_r						= thrust::raw_pointer_cast( &(ghostTagsP[0]) );
	hybridTagsUV_r						= thrust::raw_pointer_cast( &(hybridTagsUV[0]) );
	hybridTagsP_r						= thrust::raw_pointer_cast( &(hybridTagsP[0]) );
	hybridTagsUV2_r						= thrust::raw_pointer_cast( &(hybridTagsUV2[0]) );

	pressureStar_r						= thrust::raw_pointer_cast( &(pressureStar[0]) );
	ustar_r								= thrust::raw_pointer_cast( &(ustar[0]) );
	body_intercept_x_r					= thrust::raw_pointer_cast( &(body_intercept_x[0]) );
	body_intercept_y_r					= thrust::raw_pointer_cast( &(body_intercept_y[0]) );
	image_point_x_r						= thrust::raw_pointer_cast( &(image_point_x[0]) );
	image_point_y_r						= thrust::raw_pointer_cast( &(image_point_y[0]) );
	body_intercept_p_x_r				= thrust::raw_pointer_cast( &(body_intercept_p_x[0]) );
	body_intercept_p_y_r				= thrust::raw_pointer_cast( &(body_intercept_p_y[0]) );
	body_intercept_p_r					= thrust::raw_pointer_cast( &(body_intercept_p[0]) );
	image_point_p_x_r					= thrust::raw_pointer_cast( &(image_point_p_x[0]) );
	image_point_p_y_r					= thrust::raw_pointer_cast( &(image_point_p_y[0]) );
	distance_from_intersection_to_node_r= thrust::raw_pointer_cast( &(distance_from_intersection_to_node[0]) );
	distance_between_nodes_at_IB_r		= thrust::raw_pointer_cast( &(distance_between_nodes_at_IB[0]) );
	distance_from_u_to_body_r			= thrust::raw_pointer_cast( &(distance_from_u_to_body[0]) );
	distance_from_v_to_body_r			= thrust::raw_pointer_cast( &(distance_from_v_to_body[0]) );
	uv_r								= thrust::raw_pointer_cast( &(uv[0]) );

	x1_ip_r				= thrust::raw_pointer_cast( &(x1_ip[0]) );
	x2_ip_r				= thrust::raw_pointer_cast( &(x2_ip[0]) );
	y1_ip_r				= thrust::raw_pointer_cast( &(y1_ip[0]) );
	y2_ip_r				= thrust::raw_pointer_cast( &(y2_ip[0]) );
	x1_ip_p_r			= thrust::raw_pointer_cast( &(x1_ip_p[0]) );
	x2_ip_p_r			= thrust::raw_pointer_cast( &(x2_ip_p[0]) );
	y1_ip_p_r			= thrust::raw_pointer_cast( &(y1_ip_p[0]) );
	y2_ip_p_r			= thrust::raw_pointer_cast( &(y2_ip_p[0]) );
	image_point_u_r		= thrust::raw_pointer_cast( &(image_point_u[0]) );
	x1_r				= thrust::raw_pointer_cast( &(x1[0]) );
	x2_r				= thrust::raw_pointer_cast( &(x2[0]) );
	x3_r				= thrust::raw_pointer_cast( &(x3[0]) );
	x4_r				= thrust::raw_pointer_cast( &(x4[0]) );
	y1_r				= thrust::raw_pointer_cast( &(y1[0]) );
	y2_r				= thrust::raw_pointer_cast( &(y2[0]) );
	y3_r				= thrust::raw_pointer_cast( &(y3[0]) );
	y4_r				= thrust::raw_pointer_cast( &(y4[0]) );
	q1_r				= thrust::raw_pointer_cast( &(q1[0]) );
	q2_r				= thrust::raw_pointer_cast( &(q2[0]) );
	q3_r				= thrust::raw_pointer_cast( &(q3[0]) );
	q4_r				= thrust::raw_pointer_cast( &(q4[0]) );
	x1_p_r				= thrust::raw_pointer_cast( &(x1_p[0]) );
	x2_p_r				= thrust::raw_pointer_cast( &(x2_p[0]) );
	x3_p_r				= thrust::raw_pointer_cast( &(x3_p[0]) );
	x4_p_r				= thrust::raw_pointer_cast( &(x4_p[0]) );
	y1_p_r				= thrust::raw_pointer_cast( &(y1_p[0]) );
	y2_p_r				= thrust::raw_pointer_cast( &(y2_p[0]) );
	y3_p_r				= thrust::raw_pointer_cast( &(y3_p[0]) );
	y4_p_r				= thrust::raw_pointer_cast( &(y4_p[0]) );
	q1_p_r				= thrust::raw_pointer_cast( &(q1_p[0]) );
	q2_p_r				= thrust::raw_pointer_cast( &(q2_p[0]) );
	q3_p_r				= thrust::raw_pointer_cast( &(q3_p[0]) );
	q4_p_r				= thrust::raw_pointer_cast( &(q4_p[0]) );
	a0_r				= thrust::raw_pointer_cast( &(a0[0]) );
	a1_r				= thrust::raw_pointer_cast( &(a1[0]) );
	a2_r				= thrust::raw_pointer_cast( &(a2[0]) );
	a3_r				= thrust::raw_pointer_cast( &(a3[0]) );
	dudt_r				= thrust::raw_pointer_cast( &(dudt[0]) );
	ududx_r				= thrust::raw_pointer_cast( &(ududx[0]) );
	vdudy_r				= thrust::raw_pointer_cast( &(vdudy[0]) );
	dvdt_r				= thrust::raw_pointer_cast( &(dvdt[0]) );
	udvdx_r				= thrust::raw_pointer_cast( &(udvdx[0]) );
	vdvdy_r				= thrust::raw_pointer_cast( &(vdvdy[0]) );
}
