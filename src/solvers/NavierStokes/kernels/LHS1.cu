#include "hip/hip_runtime.h"
#include "LHS1.h"

namespace kernels
{
__global__
void LHS_mid_X(int *row, int *col, double *val, int *tags, int *tags2, int *tagsIn, double *a, double *b, double *dx, double *dy, double dt, double nu, int nx, int ny)
{
	if (threadIdx.x + blockDim.x * blockIdx.x >= (nx-1)*ny)
		return;
	int i 	= threadIdx.x + blockDim.x * blockIdx.x,
		I	= i % (nx-1),
		J	= i / (nx-1);
	if (I == 0 || I == nx-2 || J == 0 || J == ny-1)
		return;

	//int numE = i*5;
	//			top row - corner    mid           sides    current row
	int numE = (nx-1)*4 - 2      + (J-1)*(5*(nx-1)  - 2) + I*5 - 1;

	double temp = 1;
	if( (tags[i] == -1  && tagsIn[i] == -1) || tagsIn[i] == 0)// if point isn't tagged
	{
		//EAST
		row[numE] = i;
		col[numE] = i+1;
		val[numE] = -0.5*dt*nu*(1/(dx[I+1]*(dx[I+1]+dx[I])*0.5));
		temp += 0.5*dt*nu*(1/(dx[I+1]*(dx[I+1]+dx[I])*0.5));
		numE++;

		//WEST
		row[numE] = i;
		col[numE] = i-1;
		val[numE] = -0.5*dt*nu*(1/(dx[I]*(dx[I+1]+dx[I])*0.5));
		temp += 0.5*dt*nu*(1/(dx[I]*(dx[I+1]+dx[I])*0.5));
		numE++;

		//NORTH
		row[numE] = i;
		col[numE] = i+(nx-1);
		val[numE] = -0.5*dt*nu*(1/(dy[J]*(dy[J+1]+dy[J])*0.5));
		temp += 0.5*dt*nu*(1/(dy[J]*(dy[J+1]+dy[J])*0.5));
		numE++;

		//SOUTH
		row[numE] = i;
		col[numE] = i-(nx-1);
		val[numE] = -0.5*dt*nu*(1/(dy[J]*(dy[J-1]+dy[J])*0.5));
		temp += 0.5*dt*nu*(1/(dy[J]*(dy[J-1]+dy[J])*0.5));
		numE++;

		//CENTER
		row[numE] = i;
		col[numE] = i;
		val[numE] = temp;
		numE++;
	}
	//end untagged
	else if (tags[i]!=-1) //if point is tagged
	{
		//ADJACENT POINT
		if (tags[i] == tags2[i] - 1) // right is away from surface
		{
			row[numE] = i;
			col[numE] = i - 1;
			val[numE] = 0;
			numE ++;

			row[numE] = i;
			col[numE] = i - (nx-1);
			val[numE] = 0;
			numE ++;

			row[numE] = i;
			col[numE] = i + (nx-1);
			val[numE] = 0;
			numE ++;

			col[numE] = i + 1;
		}
		else if (tags[i] == tags2[i] + 1) //left is away from surface
		{
			row[numE] = i;
			col[numE] = i + 1;
			val[numE] = 0;
			numE ++;

			row[numE] = i;
			col[numE] = i - (nx-1);
			val[numE] = 0;
			numE ++;

			row[numE] = i;
			col[numE] = i + (nx-1);
			val[numE] = 0;
			numE ++;

			col[numE] = i - 1;
		}
		else if (tags[i] == tags2[i] + (nx-1)) // below is away from surface
		{
			row[numE] = i;
			col[numE] = i - 1;
			val[numE] = 0;
			numE ++;

			row[numE] = i;
			col[numE] = i + 1;
			val[numE] = 0;
			numE ++;

			row[numE] = i;
			col[numE] = i + (nx-1);
			val[numE] = 0;
			numE ++;

			col[numE] = i - (nx-1);
		}
		else if (tags[i] == tags2[i] - (nx-1)) // above is away from surface
		{
			row[numE] = i;
			col[numE] = i - 1;
			val[numE] = 0;
			numE ++;

			row[numE] = i;
			col[numE] = i + 1;
			val[numE] = 0;
			numE ++;

			row[numE] = i;
			col[numE] = i - (nx-1);
			val[numE] = 0;
			numE ++;

			col[numE] = i + (nx-1);
		}



		row[numE] = i;
		val[numE] = -a[i]/(a[i]+b[i]);
		numE++;

		//CENTER
		row[numE] = i;
		col[numE] = i;
		val[numE] = 1;
		numE++;
	}
	//end tagged
	else if (tagsIn[i] > 0) //inner point is tagged Note:: dx and dy must be uniform in a section with a body...
	{
		//ADJACENT POINT
		if (tags[i+1] != -1)// go right to body
		{
			row[numE] = i;
			col[numE] = i + 1;
			val[numE] = 0;
			numE ++;

			row[numE] = i;
			col[numE] = i - (nx-1);
			val[numE] = 0;
			numE ++;

			row[numE] = i;
			col[numE] = i + (nx-1);
			val[numE] = 0;
			numE ++;

			col[numE] = i-1;
			val[numE] = -(dx[I]-a[i+1]) / (dx[I] - a[i+1] + b[i+1]);
		}
		else if (tags[i-1] != -1) //go left to body
		{
			row[numE] = i;
			col[numE] = i - 1;
			val[numE] = 0;
			numE ++;

			row[numE] = i;
			col[numE] = i - (nx-1);
			val[numE] = 0;
			numE ++;

			row[numE] = i;
			col[numE] = i + (nx-1);
			val[numE] = 0;
			numE ++;

			col[numE] = i + 1;
			val[numE] = -(dx[I]-a[i-1]) / (dx[I] - a[i-1] + b[i-1]);
		}
		else if (tags[i+(nx-1)] != -1)//go north to body
		{
			row[numE] = i;
			col[numE] = i - 1;
			val[numE] = 0;
			numE ++;

			row[numE] = i;
			col[numE] = i + 1;
			val[numE] = 0;
			numE ++;

			row[numE] = i;
			col[numE] = i + (nx-1);
			val[numE] = 0;
			numE ++;

			col[numE] = i - (nx-1);
			val[numE] = -(dx[I]-a[i+(nx-1)]) / (dx[I] - a[i+(nx-1)] + b[i+(nx-1)]);
		}
		else if (tags[i-(nx-1)] != -1)//go south to body
		{
			row[numE] = i;
			col[numE] = i - 1;
			val[numE] = 0;
			numE ++;

			row[numE] = i;
			col[numE] = i + 1;
			val[numE] = 0;
			numE ++;

			row[numE] = i;
			col[numE] = i - (nx-1);
			val[numE] = 0;
			numE ++;

			col[numE] = i + (nx-1);
			val[numE] = -(dx[I]-a[i-(nx-1)]) / (dx[I] - a[i-(nx-1)] + b[i-(nx-1)]);
		}
		row[numE] = i;
		numE++;

		//CENTER
		col[numE] = i;
		row[numE] = i;
		val[numE] = 1;
		numE++;
	}
}

__global__
void LHS_BC_X(int *row, int *col, double *val, double *dx, double *dy, double dt, double nu, int nx, int ny)
{
	if (threadIdx.x + blockDim.x * blockIdx.x >= (nx-1)*ny)
		return;
	int i 	= threadIdx.x + blockDim.x * blockIdx.x,
		I	= i % (nx-1),
		J	= i / (nx-1);
	if (I != 0 && I != nx-2 && J != 0 && J != ny-1)
		return;

	double temp = 1;
	int numE = 0;
	if (J == 0)
	{
		numE = I*4;
		if (I != 0)
			numE -= 1;
	}
	else if (J == ny-1)
	{
		numE = (nx-1)*4 - 2 + (J-1)*(5*(nx-1)  - 2) + I*4;
		if (I != 0)
			numE-=1;
	}
	else
	{
		if (I == 0)
			numE = (nx-1)*4 - 2 + (J-1)*(5*(nx-1)  - 2) + I*5;
		else
			numE = (nx-1)*4 - 2 + (J-1)*(5*(nx-1)  - 2) + I*5 - 1;
	}

	//EAST
	if(I != nx-2)//check if on east boundary
	{
		row[numE] = i;
		col[numE] = i+1;
		val[numE] = -0.5*dt*nu*(1/(dx[I+1]*(dx[I+1]+dx[I])*0.5));
		temp += 0.5*dt*nu*(1/(dx[I+1]*(dx[I+1]+dx[I])*0.5));
		numE++;
	}
	else
	{
		temp += 0.5*dt*nu*(1/(dx[I+1]*(dx[I+1]+dx[I])*0.5));
	}

	//WEST
	if(I != 0)//check if on west boundary
	{
		row[numE] = i;
		col[numE] = i-1;
		val[numE] = -0.5*dt*nu*(1/(dx[I]*(dx[I+1]+dx[I])*0.5));
		temp += 0.5*dt*nu*(1/(dx[I]*(dx[I+1]+dx[I])*0.5));
		numE++;
	}
	else
	{
		temp += 0.5*dt*nu*(1/(dx[I]*(dx[I+1]+dx[I])*0.5));
	}

	//NORTH
	if(J != ny-1)//check if on north boundary
	{
		row[numE] = i;
		col[numE] = i+(nx-1);
		val[numE] = -0.5*dt*nu*(1/(dy[J]*(dy[J+1]+dy[J])*0.5));
		temp += 0.5*dt*nu*(1/(dy[J]*(dy[J+1]+dy[J])*0.5));
		numE++;
	}
	else
	{
		temp += 0.5*dt*nu*(1/(dy[J]*(dy[J]+dy[J])*0.5));
	}

	//SOUTH
	if(J != 0)//check if on south boundary
	{
		row[numE] = i;
		col[numE] = i-(nx-1);
		val[numE] = -0.5*dt*nu*(1/(dy[J]*(dy[J-1]+dy[J])*0.5));
		temp += 0.5*dt*nu*(1/(dy[J]*(dy[J-1]+dy[J])*0.5));
		numE++;
	}
	else
	{
		temp += 0.5*dt*nu*(1/(dy[J]*(dy[J]+dy[J])*0.5));
	}

	//CENTER
	row[numE] = i;
	col[numE] = i;
	val[numE] = temp;
	numE++;
}

__global__
void LHS_mid_Y(int *row, int *col, double *val, int *tags, int *tags2, int *tagsIn, double *a, double *b, double *dx, double *dy, double dt, double nu, int nx, int ny)
{
	if (threadIdx.x + blockDim.x * blockIdx.x >= nx*(ny-1))
		return;
	int ip 	= threadIdx.x + blockDim.x * blockIdx.x,
		I	= ip % nx,
		J	= ip / nx,
		i = ip + (nx-1)*ny;
	if (I == 0 || I == nx-1 || J == 0 || J == ny-2)
		return;

	int numE = (nx-1)*ny*5 - 2*ny-2*(nx-1)  +  nx*4-2  + (J-1)*(nx*5 - 2) + I*5 - 1;
	double temp = 1;

	if((tags[i] == -1 && tagsIn[i] == -1)||tagsIn[i] == 0)	//if not tagged
	{
		//EAST
		row[numE] = i;
		col[numE] = i+1;
		val[numE] = -0.5*dt*nu*(1/(dx[I]*(dx[I]+dx[I+1])*0.5));
		temp += 0.5*dt*nu*(1/(dx[I]*(dx[I]+dx[I+1])*0.5));
		numE++;

		//WEST
		row[numE] = i;
		col[numE] = i-1;
		val[numE] = -0.5*dt*nu*(1/(dx[I]*(dx[I]+dx[I-1])*0.5));
		temp += 0.5*dt*nu*(1/(dx[I]*(dx[I]+dx[I-1])*0.5));
		numE++;

		//NORTH
		row[numE] = i;
		col[numE] = i + nx;
		val[numE] = -0.5*dt*nu*(1/(dy[J+1]*(dy[J]+dy[J+1])*0.5));
		temp += 0.5*dt*nu*(1/(dy[J+1]*(dy[J]+dy[J+1])*0.5));
		numE++;

		//SOUTH
		row[numE] = i;
		col[numE] = i-nx;
		val[numE] = -0.5*dt*nu*(1/(dy[J]*(dy[J]+dy[J+1])*0.5));
		temp += 0.5*dt*nu*(1/(dy[J]*(dy[J]+dy[J+1])*0.5));
		numE++;

		//CENTER
		row[numE] = i;
		col[numE] = i;
		val[numE] = temp;
		numE++;
	}
	//end untagged section
	else if (tags[i]>0) //if point is tagged
	{
		if (tags[i] == tags2[i] - 1) //right is away from surface
		{
			row[numE] = i;
			col[numE] = i - 1;
			val[numE] = 0;
			numE++;

			row[numE] = i;
			col[numE] = i - nx;
			val[numE] = 0;
			numE++;

			row[numE] = i;
			col[numE] = 1 + nx;
			val[numE] = 0;
			numE++;

			col[numE] = i + 1;
		}
		else if (tags[i] == tags2[i] + 1)//left  is away from surface
		{
			row[numE] = i;
			col[numE] = i + 1;
			val[numE] = 0;
			numE++;

			row[numE] = i;
			col[numE] = i - nx;
			val[numE] = 0;
			numE++;

			row[numE] = i;
			col[numE] = 1 + nx;
			val[numE] = 0;
			numE++;

			col[numE] = i - 1;
		}
		else if (tags[i] == tags2[i] + nx)//below is away from surface
		{
			row[numE] = i;
			col[numE] = i - 1;
			val[numE] = 0;
			numE++;

			row[numE] = i;
			col[numE] = i + 1;
			val[numE] = 0;
			numE++;

			row[numE] = i;
			col[numE] = 1 + nx;
			val[numE] = 0;
			numE++;

			col[numE] = i - nx;
		}
		else if (tags[i] == tags2[i] - nx)//above is away from surface
		{
			row[numE] = i;
			col[numE] = i - 1;
			val[numE] = 0;
			numE++;

			row[numE] = i;
			col[numE] = i + 1;
			val[numE] = 0;
			numE++;

			row[numE] = i;
			col[numE] = 1 - nx;
			val[numE] = 0;
			numE++;

			col[numE] = i + nx;
		}
		row[numE] = i;
		val[numE] = -a[i]/(b[i]+a[i]);
		numE++;

		row[numE] = i;
		col[numE] = i;
		val[numE] = 1;
		numE++;
	}//end tagged

	else if (tagsIn[i] != -1) //inner point is tagged Note:: dx and dy must be uniform in a section with a body...
	{
		//setup adjacent point
		if (tags[i+1] != -1)// go right to body
		{
			row[numE] = i;
			col[numE] = i + 1;
			val[numE] = 0;
			numE++;

			row[numE] = i;
			col[numE] = i - nx;
			val[numE] = 0;
			numE++;

			row[numE] = i;
			col[numE] = 1 + nx;
			val[numE] = 0;
			numE++;

			col[numE] = i-1;
			val[numE] = -(dx[I]-a[i+1]) / (dx[I] - a[i+1] + b[i+1]);
		}
		else if (tags[i-1] != -1) //go left to body
		{
			row[numE] = i;
			col[numE] = i - 1;
			val[numE] = 0;
			numE++;

			row[numE] = i;
			col[numE] = i - nx;
			val[numE] = 0;
			numE++;

			row[numE] = i;
			col[numE] = 1 + nx;
			val[numE] = 0;
			numE++;

			col[numE] = i + 1;
			val[numE] = -(dx[I]-a[i-1]) / (dx[I] - a[i-1] + b[i-1]);
		}
		else if (tags[i+nx] != -1)//go north to body
		{
			row[numE] = i;
			col[numE] = i - 1;
			val[numE] = 0;
			numE++;

			row[numE] = i;
			col[numE] = i + 1;
			val[numE] = 0;
			numE++;

			row[numE] = i;
			col[numE] = 1 + nx;
			val[numE] = 0;
			numE++;

			col[numE] = i - nx;
			val[numE] = -(dx[I]-a[i+nx]) / (dx[I] - a[i+nx] + b[i+nx]);
		}
		else if (tags[i-nx] != -1)//go south to body
		{
			row[numE] = i;
			col[numE] = i - 1;
			val[numE] = 0;
			numE++;

			row[numE] = i;
			col[numE] = i + 1;
			val[numE] = 0;
			numE++;

			row[numE] = i;
			col[numE] = 1 - nx;
			val[numE] = 0;
			numE++;

			col[numE] = i + nx;
			val[numE] = -(dx[I]-a[i-nx]) / (dx[I] - a[i-nx] + b[i-nx]);
		}
		row[numE] = i;
		numE++;

		//setup p
		col[numE] = i;
		row[numE] = i;
		val[numE] = 1;
		numE++;
	}//end inside tag
}

__global__
void LHS_BC_Y(int *row, int *col, double *val, double *dx, double *dy, double dt, double nu, int nx, int ny)
{
	if (threadIdx.x + blockDim.x * blockIdx.x >= nx*(ny-1))
		return;
	int ip 	= threadIdx.x + blockDim.x * blockIdx.x,
		I	= ip % nx,
		J	= ip / nx,
		i = ip + (nx-1)*ny;
	if (I != 0 && I != nx-1 && J != 0 && J != ny-2)
		return;

	int numE = (nx-1)*ny*5 - 2*ny-2*(nx-1);
	if (J == 0)
	{
		numE += I*4;
		if (I != 0)
			numE -= 1;
	}
	else if (J == ny-2)
	{
		numE += nx*4 - 2 + (J-1)*(5*nx - 2) + I*4;
		if (I != 0)
			numE-=1;
	}
	else
	{
		if (I == 0)
			numE += nx*4 - 2 + (J-1)*(5*nx - 2) + I*5;
		else
			numE += nx*4 - 2 + (J-1)*(5*nx - 2) + I*5 - 1;
	}
	double temp = 1;

	//EAST
	if(I != nx-1)//check if on east boundary
	{
		row[numE] = i;
		col[numE] = i+1;
		val[numE] = -0.5*dt*nu*(1/(dx[I]*(dx[I]+dx[I+1])*0.5));
		temp += 0.5*dt*nu*(1/(dx[I]*(dx[I]+dx[I+1])*0.5));
		numE++;
	}
	else
	{
		temp += 0.5*dt*nu*(1/(dx[I]*(dx[I]+dx[I])*0.5));
	}

	//WEST
	if(I != 0)//check if  on west boundary
	{
		row[numE] = i;
		col[numE] = i-1;
		val[numE] = -0.5*dt*nu*(1/(dx[I]*(dx[I]+dx[I-1])*0.5));
		temp += 0.5*dt*nu*(1/(dx[I]*(dx[I]+dx[I-1])*0.5));
		numE++;
	}
	else
	{
		temp += 0.5*dt*nu*(1/(dx[I]*(dx[I]+dx[I])*0.5));
	}

	//NORTH
	if(J != ny-2)//check if on north boundary
	{
		row[numE] = i;
		col[numE] = i + nx;
		val[numE] = -0.5*dt*nu*(1/(dy[J+1]*(dy[J]+dy[J+1])*0.5));
		temp += 0.5*dt*nu*(1/(dy[J+1]*(dy[J]+dy[J+1])*0.5));
		numE++;
	}
	else
	{
		temp += 0.5*dt*nu*(1/(dy[J+1]*(dy[J]+dy[J+1])*0.5));
	}

	//SOUTH
	if(J != 0)//check if on south boundary
	{
		row[numE] = i;
		col[numE] = i-nx;
		val[numE] = -0.5*dt*nu*(1/(dy[J]*(dy[J]+dy[J+1])*0.5));
		temp += 0.5*dt*nu*(1/(dy[J]*(dy[J]+dy[J+1])*0.5));
		numE++;
	}
	else
	{
		temp += 0.5*dt*nu*(1/(dy[J]*(dy[J]+dy[J+1])*0.5));
	}

	//CENTER
	row[numE] = i;
	col[numE] = i;
	val[numE] = temp;
	numE++;
}

__global__
void LHS_mid_X_nobody(int *row, int *col, double *val, double *dx, double *dy, double dt, double nu, int nx, int ny)
{
	if (threadIdx.x + blockDim.x * blockIdx.x >= (nx-1)*ny)
		return;
	int i 	= threadIdx.x + blockDim.x * blockIdx.x,
		I	= i % (nx-1),
		J	= i / (nx-1);
	if (I == 0 || I == nx-2 || J == 0 || J == ny-1)
		return;

	//			top row - corner    mid           sides    current row
	int numE = (nx-1)*4 - 2      + (J-1)*(5*(nx-1)  - 2) + I*5 - 1;

	double temp = 1;
	//EAST
	row[numE] = i;
	col[numE] = i+1;
	val[numE] = -0.5*dt*nu*(1/(dx[I+1]*(dx[I+1]+dx[I])*0.5));
	temp += 0.5*dt*nu*(1/(dx[I+1]*(dx[I+1]+dx[I])*0.5));
	numE++;

	//WEST
	row[numE] = i;
	col[numE] = i-1;
	val[numE] = -0.5*dt*nu*(1/(dx[I]*(dx[I+1]+dx[I])*0.5));
	temp += 0.5*dt*nu*(1/(dx[I]*(dx[I+1]+dx[I])*0.5));
	numE++;

	//NORTH
	row[numE] = i;
	col[numE] = i+(nx-1);
	val[numE] = -0.5*dt*nu*(1/(dy[J]*(dy[J+1]+dy[J])*0.5));
	temp += 0.5*dt*nu*(1/(dy[J]*(dy[J+1]+dy[J])*0.5));
	numE++;

	//SOUTH
	row[numE] = i;
	col[numE] = i-(nx-1);
	val[numE] = -0.5*dt*nu*(1/(dy[J]*(dy[J-1]+dy[J])*0.5));
	temp += 0.5*dt*nu*(1/(dy[J]*(dy[J-1]+dy[J])*0.5));
	numE++;

	//CENTER
	row[numE] = i;
	col[numE] = i;
	val[numE] = temp;
	numE++;
}

__global__
void LHS_mid_Y_nobody(int *row, int *col, double *val, double *dx, double *dy, double dt, double nu, int nx, int ny)
{
	if (threadIdx.x + blockDim.x * blockIdx.x >= nx*(ny-1))
		return;
	int ip 	= threadIdx.x + blockDim.x * blockIdx.x,
		I	= ip % nx,
		J	= ip / nx,
		i = ip + (nx-1)*ny;
	if (I == 0 || I == nx-1 || J == 0 || J == ny-2)
		return;

	//         (              numU       )     (row1)    (rows2-before me)  (current row)
	int numE = (nx-1)*ny*5 - 2*ny-2*(nx-1)  +  nx*4-2  + (J-1)*(nx*5 - 2) + I*5 - 1;
	double temp = 1;

	//EAST
	row[numE] = i;
	col[numE] = i+1;
	val[numE] = -0.5*dt*nu*(1/(dx[I]*(dx[I]+dx[I+1])*0.5));
	temp += 0.5*dt*nu*(1/(dx[I]*(dx[I]+dx[I+1])*0.5));
	numE++;

	//WEST
	row[numE] = i;
	col[numE] = i-1;
	val[numE] = -0.5*dt*nu*(1/(dx[I]*(dx[I]+dx[I-1])*0.5));
	temp += 0.5*dt*nu*(1/(dx[I]*(dx[I]+dx[I-1])*0.5));
	numE++;

	//NORTH
	row[numE] = i;
	col[numE] = i + nx;
	val[numE] = -0.5*dt*nu*(1/(dy[J+1]*(dy[J]+dy[J+1])*0.5));
	temp += 0.5*dt*nu*(1/(dy[J+1]*(dy[J]+dy[J+1])*0.5));
	numE++;

	//SOUTH
	row[numE] = i;
	col[numE] = i-nx;
	val[numE] = -0.5*dt*nu*(1/(dy[J]*(dy[J]+dy[J+1])*0.5));
	temp += 0.5*dt*nu*(1/(dy[J]*(dy[J]+dy[J+1])*0.5));
	numE++;

	//CENTER
	row[numE] = i;
	col[numE] = i;
	val[numE] = temp;
	numE++;
}
}
