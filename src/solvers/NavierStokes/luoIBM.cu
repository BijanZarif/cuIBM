#include "hip/hip_runtime.h"
/***************************************************************************//**
 * \file  luoIBM.cu
 * \author Christopher Minar (minarc@oregonstate.edu)
 * \based on code by Anush Krishnan (anush@bu.edu)
 * \brief Declaration of the class oscCylinder.
 */

#include "luoIBM.h"
#include <sys/stat.h>

/**
 * \brief Constructor. Copies the database and information about the computational grid.
 *
 * \param pDB database that contains all the simulation parameters
 * \param dInfo information related to the computational grid
 */
luoIBM::luoIBM(parameterDB *pDB, domain *dInfo)
{
	paramDB = pDB;
	domInfo = dInfo;
}

/*
 * Initialise the solver
 */
void luoIBM::initialise()
{

	NavierStokesSolver::initialiseNoBody();
	NavierStokesSolver::logger.startTimer("initialise");

	int nx = NavierStokesSolver::domInfo->nx,
		ny = NavierStokesSolver::domInfo->ny;

	int numUV = (nx-1)*ny + nx*(ny-1);
	int numP  = nx*ny;
	////////////////////////////////////////////////////////////////////////////////////////////////
	//ARRAYS
	////////////////////////////////////////////////////////////////////////////////////////////////
	pressureStar.resize(numP);
	ustar.resize(numUV);
	ghostTagsUV.resize(numUV);
	hybridTagsUV.resize(numUV);
	hybridTagsUV2.resize(numUV);
	body_intercept_x.resize(numUV);
	body_intercept_y.resize(numUV);
	image_point_x.resize(numUV);
	image_point_y.resize(numUV);
	body_intercept_p_x.resize(numP);
	body_intercept_p_y.resize(numP);
	image_point_p_x.resize(numP);
	image_point_p_y.resize(numP);
	distance_from_intersection_to_node.resize(numUV);
	distance_between_nodes_at_IB.resize(numUV);
	uv.resize(numUV);

	//testing
	x1_ip.resize(numUV);
	x2_ip.resize(numUV);
	y1_ip.resize(numUV);
	y2_ip.resize(numUV);
	x1_ip_p.resize(numP);
	x2_ip_p.resize(numP);
	y1_ip_p.resize(numP);
	y2_ip_p.resize(numP);
	ip_u.resize(numUV);
	x1.resize(numUV);
	x2.resize(numUV);
	x3.resize(numUV);
	x4.resize(numUV);
	y1.resize(numUV);
	y2.resize(numUV);
	y3.resize(numUV);
	y4.resize(numUV);
	q1.resize(numUV);
	q2.resize(numUV);
	q3.resize(numUV);
	q4.resize(numUV);
	x1_p.resize(numP);
	x2_p.resize(numP);
	x3_p.resize(numP);
	x4_p.resize(numP);
	y1_p.resize(numP);
	y2_p.resize(numP);
	y3_p.resize(numP);
	y4_p.resize(numP);
	q1_p.resize(numP);
	q2_p.resize(numP);
	q3_p.resize(numP);
	q4_p.resize(numP);
	a0.resize(numP);
	a1.resize(numP);
	a2.resize(numP);
	a3.resize(numP);

	//tagpoints, size nump
	ghostTagsP.resize(numP);
	hybridTagsP.resize(numP);
	distance_from_u_to_body.resize(numP);
	distance_from_v_to_body.resize(numP);

	////////////////////////////////////////////////////////////////////////////////////////////////
	//Initialize Bodies
	////////////////////////////////////////////////////////////////////////////////////////////////
	B.initialise((*paramDB), *domInfo);
	std::cout << "Initialised bodies!" << std::endl;

	/////////////////////////////////////////////////////////////////////////////////////////////////
	//TAG POINTS
	/////////////////////////////////////////////////////////////////////////////////////////////////
	tagPoints();
	std::cout << "Tagged points!" << std::endl;

	/////////////////////////////////////////////////////////////////////////////////////////////////
	//LHS
	/////////////////////////////////////////////////////////////////////////////////////////////////
	initialiseLHS();

	/////////////////////////////////////////////////////////////////////////////////////////////////
	//OUTPUT
	/////////////////////////////////////////////////////////////////////////////////////////////////
	parameterDB  &db = *NavierStokesSolver::paramDB;
	std::string folder = db["inputs"]["caseFolder"].get<std::string>();
	std::stringstream out;
	out << folder << "/forces";
	forceFile.open(out.str().c_str());

	logger.stopTimer("initialise");
}

/*
 * Initialise the LHS matricies
 */
void luoIBM::initialiseLHS()
{
	parameterDB  &db = *NavierStokesSolver::paramDB;
	generateLHS1();
	generateLHS2();

	NavierStokesSolver::PC.generate(NavierStokesSolver::LHS1,NavierStokesSolver::LHS2, db["velocitySolve"]["preconditioner"].get<preconditionerType>(), db["PoissonSolve"]["preconditioner"].get<preconditionerType>());
	std::cout << "Assembled LUO LHS matrices!" << std::endl;
}

/**
 * \brief Writes data into files.
 */
void luoIBM::writeData()
{
	parameterDB  &db = *NavierStokesSolver::paramDB;
	double dt  = db["simulation"]["dt"].get<double>();

	logger.startTimer("output");

	writeCommon();
	calculateForce();
	if (NavierStokesSolver::timeStep == 0)
		forceFile<<"timestep\tFx\tFxX\tFxY\tFxU\tFy\n";
	forceFile << timeStep*dt << '\t' << B.forceX[0] << '\t'<<fxx<<"\t"<<fxy<<"\t"<<fxu<<"\t" << B.forceY[0] << std::endl;

	logger.stopTimer("output");
}

/**
 * \brief Writes numerical solution at current time-step,
 *        as well as the number of iterations performed in each solver.
 */
void luoIBM::writeCommon()
{
	NavierStokesSolver::writeCommon();
	parameterDB  &db = *NavierStokesSolver::paramDB;
	int nsave = db["simulation"]["nsave"].get<int>();
	std::string folder = db["inputs"]["caseFolder"].get<std::string>();

	// write body output
	if (timeStep % nsave == 0)
	{
		B.writeToFile(folder, NavierStokesSolver::timeStep);
	}

	// write the number of iterations for each solve
	iterationsFile << timeStep << '\t' << iterationCount1 << '\t' << iterationCount2 << std::endl;
}

/**
 * \brief Calculates the variables at the next time step.
 */
void luoIBM::stepTime()
{
	generateRHS1();
	solveIntermediateVelocity();
	weightUhat();
	//arrayprint(uhat,"uhat final","x");

	generateRHS2();
	solvePoisson();
	arrayprint(pressure,"pressure before","p");
	weightPressure();
	arrayprint(pressure,"pressure after","p");

	velocityProjection();

	std::cout<<timeStep<<std::endl;
	timeStep++;
	if (timeStep == 1000)
		arrayprint(u,"u","x");
}

/**
 * \brief Prints timing information and closes the different files.
 */
void luoIBM::shutDown()
{
	NavierStokesSolver::shutDown();
	forceFile.close();
}

#include "luoIBM/intermediateVelocity.inl"
#include "luoIBM/intermediatePressure.inl"
#include "luoIBM/projectVelocity.inl"
#include "luoIBM/tagpoints.inl"
#include "luoIBM/calculateForce.inl"
