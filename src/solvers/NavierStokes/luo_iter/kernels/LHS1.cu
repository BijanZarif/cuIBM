#include "hip/hip_runtime.h"
/***************************************************************************//**
 * \file LHS1.cu
 * \author Christopher Minar (minarc@oregonstate.edu)
 * \brief kernels to generate the left hand side for the intermediate velocity solve
 */

#include "LHS1.h"

namespace kernels
{
__global__
void LHS1_mid_iter_X(int *row, int *col, double *val, double *dx, double *dy, double dt, double nu, int nx, int ny,
					int *hybridTagsUV, int *ghostTagsUV, int *ns_rhs, int *interp_rhs, int *count,
					int *index1, int *index2, int *index3, int *index4,
					double *xu, double *yu, double *detA, double *alpha,
					double *b11, double *b12, double *b13, double *b14,
					double *b21, double *b22, double *b23, double *b24,
					double *b31, double *b32, double *b33, double *b34,
					double *b41, double *b42, double *b43, double *b44,
					double *q1, double *q2, double *q3, double *q4
					)
{
	if (threadIdx.x + blockDim.x * blockIdx.x >= (nx-1)*ny)
		return;
	int iu 	= threadIdx.x + blockDim.x * blockIdx.x,
		I	= iu % (nx-1),
		J	= iu / (nx-1);
	if (I == 0 || I == nx-2 || J == 0 || J == ny-1)
		return;

	//int numE = i*5;
	//			top row - corner    mid           sides    current row
	int numE = (nx-1)*4 - 2      + (J-1)*(5*(nx-1)  - 2) + I*5 - 1;

	double temp = 1;

	if (hybridTagsUV[iu]>0)
	{
		int interp_index[4] = {index1[iu], index2[iu], index3[iu], index4[iu]};
		int ns_index[5] = {iu + (nx-1), iu + 1, iu - (nx-1), iu -1, iu}; //n e s w p
		double q[4] = {q1[iu], q2[iu], q3[iu], q4[iu]};
		double CInterp[4];
		double Cns[5];
		double	x=xu[I],
				y=yu[J];
		Cns[0] = -dt*nu/(dy[J+1]*(dy[J]+dy[J+1]));
		Cns[1] = -dt*nu/(dx[I]  *(dx[I]+dx[I+1]));
		Cns[2] = -dt*nu/(dy[J]  *(dy[J]+dy[J+1]));
		Cns[3] = -dt*nu/(dx[I]  *(dx[I]+dx[I-1]));
		Cns[4] = -Cns[0] - Cns[1] - Cns[2] - Cns[3];
		CInterp[0] = (b11[iu] + b21[iu]*x + b31[iu]*y + b41[iu]*x*y)/detA[iu];
		CInterp[1] = (b12[iu] + b22[iu]*x + b32[iu]*y + b42[iu]*x*y)/detA[iu];
		CInterp[2] = (b13[iu] + b23[iu]*x + b33[iu]*y + b43[iu]*x*y)/detA[iu];
		CInterp[3] = (b14[iu] + b24[iu]*x + b34[iu]*y + b44[iu]*x*y)/detA[iu];
		for (int l=0; l<4; l++)
		{
			Cns[l] = Cns[l]*(1-alpha[iu])/Cns[4];
			CInterp[l] = CInterp[l]*alpha[iu];
		}
		/*   0  1  2		NW  N   NE
		 *   3  4  5		W   P   E
		 *   6  7  8		SW  S   SE
		 */
		int stencil_index[9]    = {iu + (nx-1) - 1, iu + (nx-1), iu + (nx-1) + 1,
								   iu - 1         , iu         , iu + 1,
								   iu - (nx-1) - 1, iu - (nx-1), iu - (nx-1) + 1};
		double stencil[9] = {0, Cns[0], 0, Cns[3], 1, Cns[1], 0, Cns[2], 0};
		//combine ns and interp stencils
		bool stencil_used[9] = {false, true, false, true, true, true, false, true, false};
		for (int n=0;n<4;n++)
		{
			for (int m=0;m<9;m++)
			{
				if (stencil_index[m] == interp_index[n] && m != 4)
				{
					stencil[m] += CInterp[n]; //flag should this be minus?
				}
			}
		}
		//add ns to sparse matrix
		for (int m = 0; m<9; m++)
		{
			if (stencil_used[m])
			{
				row[numE] = iu;
				col[numE] = stencil_index[m];
				val[numE] = stencil[m];
				numE++;
			}
		}
		ns_rhs[iu] = (1-alpha[iu])/Cns[4];
		interp_rhs[iu] = 0;
		//calc new numE
		numE = ny*(nx-1) + ny*2 + (nx-1)*2    +   nx*(ny-1) + nx*2 + (ny-1)*2 + count[iu];
		//add interp corner to sparse matrix
		for (int n=0;n<4;n++)
		{
			for (int m=0;m<9;m++)
			{
				if (stencil_index[m] == interp_index[n] && !stencil_used[m])
				{
					row[numE] = iu;
					col[numE] = interp_index[n];
					val[numE] = CInterp[n];
				}
				else if(stencil_index[m] == interp_index[n] && stencil_used[m])
					interp_rhs[iu] += CInterp[n]*q[n];
			}
		}
	}
	else if (ghostTagsUV[iu]>0)
	{

	}
	else
	{
	temp = 0.5*dt*nu*(1/(dx[I+1]*(dx[I+1]+dx[I])*0.5)) + 0.5*dt*nu*(1/(dx[I]*(dx[I+1]+dx[I])*0.5)) + 0.5*dt*nu*(1/(dy[J]*(dy[J+1]+dy[J])*0.5)) + 0.5*dt*nu*(1/(dy[J]*(dy[J-1]+dy[J])*0.5));
	//EAST
	row[numE] = iu;
	col[numE] = iu+1;
	val[numE] = -0.5*dt*nu*(1/(dx[I+1]*(dx[I+1]+dx[I])*0.5))/temp;
	numE++;

	//WEST
	row[numE] = iu;
	col[numE] = iu-1;
	val[numE] = -0.5*dt*nu*(1/(dx[I]*(dx[I+1]+dx[I])*0.5))/temp;
	numE++;

	//NORTH
	row[numE] = iu;
	col[numE] = iu+(nx-1);
	val[numE] = -0.5*dt*nu*(1/(dy[J]*(dy[J+1]+dy[J])*0.5))/temp;
	numE++;

	//SOUTH
	row[numE] = iu;
	col[numE] = iu-(nx-1);
	val[numE] = -0.5*dt*nu*(1/(dy[J]*(dy[J-1]+dy[J])*0.5))/temp;
	numE++;

	//CENTER
	row[numE] = iu;
	col[numE] = iu;
	val[numE] = 1;
	numE++;
	ns_rhs[iu] = 1/temp;
	interp_rhs[iu] = 0;
	}
}

__global__
void LHS1_mid_iter_Y(int *row, int *col, double *val, int *ghostTagsUV, double *dx, double *dy, double dt, double nu, int nx, int ny)
{
	if (threadIdx.x + blockDim.x * blockIdx.x >= nx*(ny-1))
		return;
	int ip 	= threadIdx.x + blockDim.x * blockIdx.x,
		I	= ip % nx,
		J	= ip / nx,
		i = ip + (nx-1)*ny;
	if (I == 0 || I == nx-1 || J == 0 || J == ny-2)
		return;

	int numE = (nx-1)*ny*5 - 2*ny-2*(nx-1)  +  nx*4-2  + (J-1)*(nx*5 - 2) + I*5 - 1;
	double temp = 1;

	//EAST
	row[numE] = i;
	col[numE] = i+1;
	val[numE] = -0.5*dt*nu*(1/(dx[I]*(dx[I]+dx[I+1])*0.5));
	temp += 0.5*dt*nu*(1/(dx[I]*(dx[I]+dx[I+1])*0.5));
	numE++;

	//WEST
	row[numE] = i;
	col[numE] = i-1;
	val[numE] = -0.5*dt*nu*(1/(dx[I]*(dx[I]+dx[I-1])*0.5));
	temp += 0.5*dt*nu*(1/(dx[I]*(dx[I]+dx[I-1])*0.5));
	numE++;

	//NORTH
	row[numE] = i;
	col[numE] = i + nx;
	val[numE] = -0.5*dt*nu*(1/(dy[J+1]*(dy[J]+dy[J+1])*0.5));
	temp += 0.5*dt*nu*(1/(dy[J+1]*(dy[J]+dy[J+1])*0.5));
	numE++;

	//SOUTH
	row[numE] = i;
	col[numE] = i-nx;
	val[numE] = -0.5*dt*nu*(1/(dy[J]*(dy[J]+dy[J+1])*0.5));
	temp += 0.5*dt*nu*(1/(dy[J]*(dy[J]+dy[J+1])*0.5));
	numE++;

	//CENTER
	row[numE] = i;
	col[numE] = i;
	val[numE] = temp;
	numE++;
}

}//end kernel
